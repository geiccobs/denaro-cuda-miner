#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdbool.h>

#include "kernel.cuh"
#include "sha256.cuh"
#include "../requests.cuh"
#include "../crypto/hex.cuh"

#define TOTAL_SIZE 108
#define MAX_SHARES 16

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define CLEAR() printf("\033[H\033[J")

__device__ __constant__ char share_chunk_c[64];
__device__ __constant__ size_t share_difficulty_c;

__device__ __forceinline__ void sha256_to_hex(unsigned char *hash, char *hex) {
    static const char digits[] = "0123456789abcdef";

#pragma unroll
    for (int i = 0; i < 16; ++i) {
        char lo_nibble = digits[hash[i] & 0x0F];
        char hi_nibble = digits[(hash[i] & 0xF0) >> 4];
        *hex++ = hi_nibble;
        *hex++ = lo_nibble;
    }
    *hex = '\0';
}

__device__ __forceinline__ bool is_valid(const char* str) {
    int mask = 0;

#pragma unroll
    for (int i = 0; i < share_difficulty_c; ++i) {
        mask |= (str[i] ^ share_chunk_c[i]);
    }
    return mask == 0;
}

__global__ void miner(unsigned char **out, bool *stop, unsigned char *prefix, int *share_id) {
    const __restrict__ uint32_t tid = threadIdx.x;

    __shared__ SHA256_CTX prefix_ctx;
    if (tid == 0) {
        sha256_init_dev(&prefix_ctx);
        sha256_update_dev(&prefix_ctx, prefix, sizeof(unsigned char) * (TOTAL_SIZE-4));
    }
    __syncthreads();

    unsigned char _hex[TOTAL_SIZE];
    memcpy(_hex, prefix, sizeof(unsigned char) * (TOTAL_SIZE-4));

    SHA256_CTX ctx;
    unsigned char hash[32];
    char hash_hex[64];

    for (uint32_t index = blockIdx.x * blockDim.x + tid; !(*stop); index += blockDim.x * gridDim.x) {
        _hex[TOTAL_SIZE-1] = index;
        _hex[TOTAL_SIZE-2] = index >> 8;
        _hex[TOTAL_SIZE-3] = index >> 16;
        _hex[TOTAL_SIZE-4] = index >> 24;

        memcpy(&ctx, &prefix_ctx, sizeof(SHA256_CTX));

        sha256_update_dev(&ctx, _hex + (TOTAL_SIZE-4), sizeof(unsigned char) * 4);
        sha256_final_dev(&ctx, hash);
        sha256_to_hex(hash, hash_hex);

        if (is_valid(hash_hex)) {
            int id = atomicAdd(share_id, 1);
            memcpy(out[id], _hex, sizeof(unsigned char) * TOTAL_SIZE);

            if (id >= MAX_SHARES-2) {
                *stop = true;
            }
        }

        if (index >= 0xFFFFFFFF) {
            *stop = true;
        }
    }
}

void start(GpuSettings *settings, ManagerData *managerData) {
    auto res = hipSetDevice(settings->deviceId);
    if (res != hipSuccess) {
        printf("Error setting device: %s\n", hipGetErrorString(res));
        return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, settings->deviceId);

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));

    // allocate memory on the device
    int zero = 0;

    bool *stop_g;
    hipMallocManaged(&stop_g, sizeof(bool));
    hipMemcpy(stop_g, &zero, sizeof(bool), hipMemcpyHostToDevice);

    int *share_id;
    hipMallocManaged(&share_id, sizeof(int));
    hipMemcpy(share_id, &zero, sizeof(int), hipMemcpyHostToDevice);

    unsigned char *prefix_g;
    hipMallocManaged(&prefix_g, sizeof(unsigned char) * (TOTAL_SIZE-4));

    unsigned char **out_g;
    hipMallocManaged(&out_g, sizeof(unsigned char*) * MAX_SHARES);

    for (int i = 0; i < MAX_SHARES; ++i) {
        hipMallocManaged(&out_g[i], sizeof(unsigned char) * TOTAL_SIZE);
        hipMemset(out_g[i], 0, sizeof(unsigned char) * TOTAL_SIZE);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(share_chunk_c), managerData->shareChunk, sizeof(char) * 64);
    hipMemcpyToSymbol(HIP_SYMBOL(share_difficulty_c), &settings->shareDifficulty, sizeof(size_t));

    size_t num_threads = settings->threads;
    if (num_threads == 0) {
        num_threads = deviceProp.maxThreadsPerBlock;
    }
    size_t num_blocks = settings->blocks;
    if (num_blocks == 0) {
        num_blocks = (deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor) / num_threads;
    }

    if (settings->verbose) printf("Starting miner with %zu blocks and %zu threads\n", num_blocks, num_threads);

    hipError_t err;
    hipEvent_t start;
    hipEvent_t end;
    uint loops_count = 0;

    err = hipEventCreate(&start);
    if (err != hipSuccess) {
        printf("Failed to create start event: %s\n", hipGetErrorString(err));
    }

    err = hipEventCreate(&end);
    if (err != hipSuccess) {
        printf("Failed to create end event: %s\n", hipGetErrorString(err));
        hipEventDestroy(start);
    }

    while (!(*managerData->stop)) {
        float elapsed_ms = 0.0f;

        err = hipEventRecord(start, 0);
        if (err != hipSuccess) {
            printf("Failed to record start event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        time_t now = time(NULL);

        hipMemcpy(prefix_g, managerData->prefix, sizeof(unsigned char) * (TOTAL_SIZE-4), hipMemcpyHostToDevice);

        prefix_g[98] = now & 0xFF;
        prefix_g[99] = (now >> 8) & 0xFF;
        prefix_g[100] = (now >> 16) & 0xFF;
        prefix_g[101] = (now >> 24) & 0xFF;

        miner<<<num_blocks,num_threads>>> (out_g, stop_g, prefix_g, share_id);
        checkCudaErrors(hipDeviceSynchronize());

        err = hipEventRecord(end, 0);
        if (err != hipSuccess) {
            printf("Failed to record end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        err = hipEventSynchronize(end);
        if (err != hipSuccess) {
            printf("Failed to synchronize end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        err = hipEventElapsedTime(&elapsed_ms, start, end);
        if (err != hipSuccess) {
            printf("Failed to get elapsed time: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        if (!settings->silent) {
            float hashrate = (pow(2, 32) - 1) / (elapsed_ms / 1000.0) / pow(10, 9);

            CLEAR();
            printf("Denaro GPU Miner\n\n");
            printf("Device: %s\n", deviceProp.name);
            printf("Threads: %zu\n", num_threads);
            printf("Blocks: %zu\n\n", num_blocks);

            printf("Node: %s\n", settings->nodeUrl);
            printf("Pool: %s\n\n", settings->poolUrl);

            printf("Accepted shares: %d\n\n", managerData->shares);

            printf("Hashrate: %.2f GH/s\n", hashrate);
        }

        if (*share_id > 0) {
            Share resp;

            unsigned char *out;
            hipMallocManaged(&out, sizeof(unsigned char) * TOTAL_SIZE);

            for (int i = 0; i < MIN(*share_id, MAX_SHARES); ++i) {
                hipMemcpy(out, out_g[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyDeviceToHost);

                if (out[0] == 2) {
                    resp = share(
                            settings->poolUrl,
                            bin2hex(out, TOTAL_SIZE),
                            managerData->miningInfo.result.pending_transactions_hashes,
                            managerData->miningInfo.result.pending_transactions_count,
                            managerData->miningInfo.result.last_block.id + 1
                    );
                    if (resp.ok) {
                        if (settings->verbose) {
                            printf("Share accepted: %s\n", bin2hex(out, TOTAL_SIZE));
                        }
                        managerData->shares++;
                    } else {
                        if (settings->verbose) {
                            printf("Share not accepted: %s\n", resp.error);
                        }
                        *managerData->stop = true;
                    }
                }
                hipMemset(out_g[i], 0, sizeof(unsigned char) * TOTAL_SIZE);
            }
            *share_id = 0;
        }

        *stop_g = false;
        loops_count++;
    }

    for (int i = 0; i < MAX_SHARES; ++i) {
        hipFree(out_g[i]);
    }
    hipFree(out_g);

    hipFree(stop_g);
    hipFree(share_id);
    hipFree(prefix_g);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceReset();
}